#include "hip/hip_runtime.h"
/********************************************************************
*  sample.cu
*  This is a example of the CUDA program.
*********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include "definitions.h"

#define float_lt(a,b) ((*a)<(*b))

__device__ void int_qsort(float *arr, unsigned n);
__device__ void torben(float m[], int n, float* med);

/// filtruje �rodek Obrazka
__global__ static void CUDAMedianMain(Matrix din, Matrix dout, Mask Okno, unsigned int startw, unsigned int starth)
{
	unsigned int r,c,rb,cb;
	float* rowin;
	float* rowout;

	int less, greater, equal;
	int n = Okno.e;
	int row,col;
	float min, max, guess, maxltguess, mingtguess, ret;

	r = blockIdx.y*blockDim.y + threadIdx.y + Okno.py + starth;	// srodek okna
	c = blockIdx.x*blockDim.x + threadIdx.x + Okno.px + startw;
	rb = r - Okno.py;	// lewy g�rny r�g okna
	cb = c - Okno.px;

	if(r>=din.height-Okno.py || c>=din.width-Okno.px)
		return;

	rowin = (float*)((char*)din.data + rb*din.pitch);
	min = max = rowin[0];
	for (row=rb ; row<rb+Okno.wy ; row++)
		for (col=cb ; col<cb+Okno.wx ; col++)
		{
			rowin = (float*)((char*)din.data + row*din.pitch);
			if (rowin[col]<min) min=rowin[col];
			if (rowin[col]>max) max=rowin[col];
		}
		while (1) {
			guess = (min+max)/2;
			less = 0; greater = 0; equal = 0;
			maxltguess = min ;
			mingtguess = max ;
			for (row=rb; row<rb+Okno.wy; row++)
				for (col=cb; col<cb+Okno.wx; col++)
				{
					rowin = (float*)((char*)din.data + row*din.pitch);
					if (rowin[col]<guess) {
						less++;
						if (rowin[col]>maxltguess) maxltguess = rowin[col] ;
					} else if (rowin[col]>guess) {
						greater++;
						if (rowin[col]<mingtguess) mingtguess = rowin[col] ;
					} else equal++;
				}
				if (less <= (n+1)/2 && greater <= (n+1)/2) break ;
				else if (less>greater) max = maxltguess ;
				else min = mingtguess;
		}
		if (less >= (n+1)/2) ret = maxltguess;
		else if (less+equal >= (n+1)/2) ret = guess;
		else ret =  mingtguess;

		rowout = (float*)((char*)dout.data + r*dout.pitch);
		rowout[c] = ret;
}

///
/// dxx, dyy - po�owa z wielko�ci nieparzystego okna - po to aby zacz�� obr�bk� w pierwszm przebiegu od tych fragment�w obrazu kt�re nie s� brzegami (mieszcz� ca�e okno)
/// startw, starth - indexy od kt�rych powinien zacz�� si� wykonywa� grid (b�d� dodawane do tych obliczonych)
void RunCUDA(Matrix din, Matrix dout, dim3 tpb, dim3 nob, Mask Okno, unsigned int startw, unsigned int starth)
{
//	CUDAMedianMain<<<nob, tpb, Okno.e*sizeof(float)>>>(din, dout, Okno, startw, starth);
	CUDAMedianMain<<<nob, tpb>>>(din, dout, Okno, startw, starth);
}



/*
__device__ void torben(float m[], int n, float* med)
{
	int i, less, greater, equal;
	float min, max, guess, maxltguess, mingtguess;
	min = max = m[0] ;
	for (i=1 ; i<n ; i++) {
		if (m[i]<min) min=m[i];
		if (m[i]>max) max=m[i];
	}
	while (1) {
		guess = (min+max)/2;
		less = 0; greater = 0; equal = 0;
		maxltguess = min ;
		mingtguess = max ;
		for (i=0; i<n; i++) {
			if (m[i]<guess) {
				less++;
				if (m[i]>maxltguess) maxltguess = m[i] ;
			} else if (m[i]>guess) {
				greater++;
				if (m[i]<mingtguess) mingtguess = m[i] ;
			} else equal++;
		}
		if (less <= (n+1)/2 && greater <= (n+1)/2) break ;
		else if (less>greater) max = maxltguess ;
		else min = mingtguess;
	}
	if (less >= (n+1)/2) *(med)= maxltguess;
	else if (less+equal >= (n+1)/2) *(med) = guess;
	else *(med) = mingtguess;
}*/
